#include <hip/hip_runtime.h>

// CUDA kernel function for computing the partial sum of the integral
template <typename T>
__global__ void IntegralKernel(T (*expression)(T), T x1, T x2, T distance, int Accurate_level, T* results) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    T local_sum = 0;

    for (int i = idx; i < Accurate_level; i += stride) {
        T x = x1 + i * distance;
        if (x < x2) {
            local_sum += distance * expression(x);
        }
    }

    results[idx] = local_sum;
}

// Example expression function to be used on the device
__device__ float deviceExpression(float x) {
    return sinf(x);
}

// Host function to call the CUDA kernel
template <typename T>
T Integral(T (*expression)(T), T x1, T x2, T Accurate_level) {
    int blockSize = 256;
    int numBlocks = (Accurate_level + blockSize - 1) / blockSize;

    T distance = (x2 - x1) / Accurate_level;
    T* results;
    hipMallocManaged(&results, numBlocks * blockSize * sizeof(T));

    IntegralKernel<<<numBlocks, blockSize>>>(expression, x1, x2, distance, Accurate_level, results);
    hipDeviceSynchronize();

    T sum = 0;
    for (int i = 0; i < numBlocks * blockSize; ++i) {
        sum += results[i];
    }

    hipFree(results);

    return sum;
}
